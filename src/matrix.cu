#include "hip/hip_runtime.h"
#include "matrix.h"
// #include <cassert>

#include <hip/hip_runtime.h>

template <typename T>
__global__ void matrixMul(T *a, T *b, T *c, int N){
    // Calculate the global row and column for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check for our matrix
    if(row < N && col < N){
        // Accumulate a partial result
        T tmp = 0;
        for(int i = 0; i < N; i++){
            tmp += a[row * N + i] * b[i * N + col];
        }

        // Write back the result
        c[row * N + col] = tmp;
    }
}

// Initializes a square matrix with random numbers between 0-100
template <typename T>
void init_matrix(T *m, int N){
    for(int i = 0; i < N * N; i++){
        m[i] = static_cast<T>(rand() % 100);
    }
}

// Verify the result on the CPU
template <typename T>
void verify_result(T *a, T *b, T *c, int N){
    int tmp;
    // For every row...
    for(int i = 0; i < N; i++){
        // For every col...
        for(int j = 0; j < N; j++){
            // For every element in the row-col pair
            tmp = 0;
            for(int k = 0; k < N; k++){
                tmp += a[i * N + k] * b[k * N + j];
            }
            
            // Check each result
            // assert(tmp == c[i * N + j]);
        }
    }
}

template <typename T>
void launch_kernel_and_profile(T *a, T *b, T *c, int N, dim3 THREADS, dim3 BLOCKS) {
    matrixMul<<<BLOCKS, THREADS>>>(a, b, c, N);
    hipDeviceSynchronize();
}

// Explicit instation of needed types
template __global__ void matrixMul<float>(float*, float*, float*, int);
template __global__ void matrixMul<double>(double*, double*, double*, int);

template void init_matrix<float>(float*, int);
template void init_matrix<double>(double*, int);

template void launch_kernel_and_profile<float>(float*, float*, float*, int, dim3, dim3);
template void launch_kernel_and_profile<double>(double*, double*, double*, int, dim3, dim3);

template void verify_result<float>(float*, float*, float*, int);
template void verify_result<double>(double*, double*, double*, int);